#include "hip/hip_runtime.h"
#include "ReadFlow.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "nvcuvid.h"
//#include "opencv2/opencv_modules.hpp"
#include "opencv2/cudev/common.hpp"

using namespace std;
using namespace cv;

#include <iostream>
#include <cmath>

using namespace std;

#define HIP_PI_F 3.141592654f

// void cartesian_to_polar(float x, float y, float& magnitude, float& angle_degrees) {
//     magnitude = sqrtf(x * x + y * y);
//     angle_degrees = atanf(y / x) * 180.0f / HIP_PI_F;
//     if (x < 0) {
//         angle_degrees += 180.0f;
//     }
//     else if (y < 0) {
//         angle_degrees += 360.0f;
//     }
// }


// int main() {
//     int lastDirection = 0;
//     int direction = 0;
//     float lastAngle = 0;
    
//     for(float x=-3.0; x<3.0; x+=0.2f) {
//         for(float y=-3.0; y<3.0; y+=0.2f) {
            
//             if(x < 0.05 && x > -0.05) {
//                 x = 0.0f;
//             }
            
//             if(y < 0.05 && y > -0.05) {
//                 y = 0.0f;
//             }
            
//             float magnitude;
//             float angle_degrees;
            
//             cartesian_to_polar(x, y, magnitude, angle_degrees);
            
//             int newDirection = 0;
//             float diff = lastAngle - angle_degrees;
//             lastAngle = angle_degrees;
            
//             if(diff < 0) {
//                 newDirection = -1;
//             }
//             if(diff > 0) {
//                 newDirection = 1;
//             }
            
//             if(newDirection != lastDirection) {
//                 lastDirection = newDirection;
//                 cout << "Dir change!" << std::endl;
//                 cout << "Angle: " << angle_degrees << " X: " << x << " Y: " << y << " Mag: " << magnitude << std::endl;
//             }
//         }
        
//     }
//     return 0;
// }

#define HIP_PI_F 3.141592654f

__device__
void cartesian_to_polar(float x, float y, float& magnitude, float& angle_degrees) {
    magnitude = sqrtf(x * x + y * y);
    angle_degrees = atanf(y / x) * 180.0f / HIP_PI_F;
    if (x < 0) {
        angle_degrees += 180.0f;
    }
    else if (y < 0) {
        angle_degrees += 360.0f;
    }
}

__global__ void MAT_POOL(
    int16_t* flowPtr,
    size_t flowPitch,
    int32_t* output,
    uint8_t pools,
    float threshold
){
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    const size_t mFlowPitch = flowPitch >> 2;

    int flowPtrAddr = (y * mFlowPitch) + (x * 2);

    float flow_x_val = (float)flowPtr[flowPtrAddr];
    float flow_y_val = (float)flowPtr[flowPtrAddr + 1];

    float magnitude;
    float angle;

    cartesian_to_polar(flow_x_val, flow_y_val, magnitude, angle);
    angle = angle / 360.0f;

    if (magnitude < threshold) {
        return;
    }

    uint8_t pool =  round(angle * (float)pools);
    
    if (pool < 0)
        pool = 0;
    
    if (pool >= pools)
        pool = pools - 1;

    ::atomicAdd((int*)output + pool, 1);
}

void runMatPool(cv::cuda::GpuMat flow, cv::cuda::GpuMat output, uint8_t pools, float threshold)
{
    assert(flow.channels() == 2);
    assert(output.rows == 1 && output.cols == pools);

    dim3 block(8, 8);
    dim3 grid(divUp(flow.cols, block.x), divUp(flow.rows, block.y));

    MAT_POOL << <grid, block >> > (
        flow.ptr<int16_t>(),
        flow.step,
        output.ptr<int32_t>(),
        pools,
        threshold
    );

    CV_CUDEV_SAFE_CALL(hipGetLastError());
}
